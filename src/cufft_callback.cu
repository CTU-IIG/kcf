#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>

__global__ void apply_window_kernel( const float *dataIn, const float* window, float* dataOut, size_t dataSize, size_t windowSize)
{
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x;
         i < dataSize;
         i += blockDim.x * gridDim.x)
    {
        dataOut[i] = dataIn[i] * window[i%windowSize];
    }
}

void cuFFT::applyWindow(MatScaleFeats &patch_feats_in, MatDynMem &window, MatScaleFeats &tmp){

    assert(patch_feats_in.total() == tmp.total());

    const size_t dataSize = patch_feats_in.total();
    const size_t windowSize = window.total();
    const dim3 threads(256);
    const dim3 blocks((dataSize + threads.x - 1) / threads.x);

    const float *featPtr = patch_feats_in.deviceMem();
    const float *windowPtr = window.deviceMem();
    float *tmpPtr = tmp.deviceMem();

    apply_window_kernel<<<blocks, threads>>>(featPtr, windowPtr, tmpPtr, dataSize, windowSize);
    CudaCheckError();
}

__global__ void scale_kernel( const float *dataIn, float* dataOut, size_t dataSize, float alpha)
{
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x;
         i < dataSize;
         i += blockDim.x * gridDim.x)
    {
        dataOut[i] = dataIn[i] * alpha;
    }
}

void cuFFT::scale(MatScales &data, float alpha){

    float *in = data.deviceMem();
    float *out = in;

    const size_t dataSize = data.total();
    const dim3 threads(256);
    const dim3 blocks((dataSize + threads.x - 1) / threads.x);

    scale_kernel<<<blocks, threads>>>(in, out, dataSize, alpha);
    CudaCheckError();
}
